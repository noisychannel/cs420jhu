#include "hip/hip_runtime.h"
/*******************************************************************************
*
*  smooth.cu
*
*  This provides a CUDA implementation of a kernel smooother.
*   http://en.wikipedia.org/wiki/Kernel_smoother
*  The particular smoother in this file is a nearest neighbor smoother
*  in order to keep the code as simple to understand as possible.
*
*  This is implemeneted for 2-d square grids.
*
*  Parameters of note:
*    dataWidth -- size of the data is dataWidth^2
*    halfWidth -- region around point x,y to smooth
*        k smooths box with corners [x-k,y-k] to [x+k,y+k]
*
*  The smoothed region is only defined for the interior that has the kernel
*   defined inside the boundary, e.g. for gridWidth=10, halfWidth=2 the
*   region from 2,2 to 7,7 will be smoothed. 
*
********************************************************************************/

/*******************************************************************************
*
*  CUDA concepts
*
*  This file shows how to use many features of CUDA:
*     2d grids
*     pitch allocation
*
********************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>

// Data is of size dataWidth * dataWidth
//const unsigned int dataWidth = 4112;

// Parameter to express the smoothing kernel halfwidth
//const unsigned int halfWidth = 8;

// Size of the CUDA threadBlock
//const unsigned int blockWidth = 16;


/* Small values good for testing */

// Data is of size dataWidth * dataWidth
const unsigned int dataWidth = 8;

// Parameter to express the smoothing kernel halfwidth
const unsigned int halfWidth = 1;

// Size of the CUDA threadBlock
const unsigned int blockWidth = 2;



/*------------------------------------------------------------------------------
* Name: NNSmoothKernel
* Action:  The CUDA kernel that implements kernel smoothing.
*             Yuck, that's two senses of kernel.
*-----------------------------------------------------------------------------*/
__global__ void NNSmoothKernel ( float* pFieldIn, float* pFieldOut, size_t pitch, unsigned int halfwidth )
{ 
  // pitch is in bytes, figure out the number of elements for addressing memory locations in pFieldIn and pFieldOut
  unsigned pitchels = pitch/sizeof(float);

  ......

} 


/*------------------------------------------------------------------------------
* Name:  SmoothField
* Action:  Host entry point to kernel smoother
*-----------------------------------------------------------------------------*/
bool SmoothField ( float* pHostFieldIn, float *pHostFieldOut ) 
{ 
  float * pDeviceFieldIn = 0;
  float * pDeviceFieldOut = 0;

  size_t pitch, pitchout;

  struct timeval ta, tb, tc, td;

  // Check the grid dimensions and extract parameters.  See top description about restrictions
  assert(((dataWidth-2*halfWidth) % blockWidth) == 0 );

  gettimeofday ( &ta, NULL );

  // Place the data set on device memory
  hipMallocPitch((void**)&pDeviceFieldIn, &pitch, dataWidth*sizeof(float), dataWidth ); 
  hipMemcpy2D ( pDeviceFieldIn, pitch,
                 pHostFieldIn, dataWidth*sizeof(float), dataWidth*sizeof(float), dataWidth,
                 hipMemcpyHostToDevice); 

  // Allocate the output
  hipMallocPitch((void**)&pDeviceFieldOut, &pitchout, dataWidth*sizeof(float), dataWidth ); 

  gettimeofday ( &tb, NULL );

  // Construct a 2d grid/block from the parameters in CUDAGrid
  const dim3 DimBlock; //.....TODO
  const dim3 DimGrid; //.....TODO

  // Invoke the kernel
  NNSmoothKernel <<<DimGrid,DimBlock>>> ( pDeviceFieldIn, pDeviceFieldOut, pitch, halfWidth ); 

  gettimeofday ( &tc, NULL );

  // Retrieve the results
  hipMemcpy2D(pHostFieldOut, dataWidth*sizeof(float), 
               pDeviceFieldOut, pitchout, dataWidth*sizeof(float), dataWidth,
               hipMemcpyDeviceToHost); 

  gettimeofday ( &td, NULL );


  if ( ta.tv_usec < td.tv_usec )
  {
    printf ("Elapsed total time (s/m): %ld:%d\n", td.tv_sec - ta.tv_sec, td.tv_usec - ta.tv_usec );
  } else {
    printf ("Elapsed total time (s/m): %ld:%d\n", td.tv_sec - ta.tv_sec - 1, 1000000 - td.tv_usec + ta.tv_usec );
  }

  if ( tb.tv_usec < tc.tv_usec )
  {
    printf ("Elapsed kernel time (s/m): %ld:%d\n", tc.tv_sec - tb.tv_sec, tc.tv_usec - tb.tv_usec );
  } else {
    printf ("Elapsed kernel time (s/m): %ld:%d\n", tc.tv_sec - tb.tv_sec - 1, 1000000 - tc.tv_usec + tb.tv_usec );
  }

  return true;
}



/*------------------------------------------------------------------------------
* Name:  initField
* Action:  Initialize a field to predictable values.
*    This is a useful format for debugging, because values 
*    accumulate to their initial value.
*-----------------------------------------------------------------------------*/
void initField ( unsigned dim, float* pField )
{
  for ( unsigned j=0; j<dim; j++ )
  {
    for ( unsigned i=0; i<dim; i++ )
    {
      pField[j*dim+i] = j + i;
    }
  }
}


/*------------------------------------------------------------------------------
* Name:  main
* Action:  Entry point
*-----------------------------------------------------------------------------*/
int main ()
{

  // Create the input field
  float *field = (float *) malloc ( dataWidth * dataWidth * sizeof(float));
  initField ( dataWidth, field );

  // Create the output field
  float *out = (float *) malloc ( dataWidth * dataWidth * sizeof(float));

  // Call the kernel
  SmoothField ( field, out );

  // Print the output field (for debugging purposes.
  for ( unsigned j=0; j< dataWidth; j++ )
  {
    for ( unsigned i=0; i< dataWidth; i++ )
    {
      if ( ( i >= halfWidth ) && 
           ( j >= halfWidth ) &&
           ( i < ( dataWidth - halfWidth )) &&
           ( j < ( dataWidth - halfWidth )) )
      {
        printf ("%4.4f, ", out[j*dataWidth + i]);
      }
      else
      {
        printf ("%4.4f, ", 0.0f );
      }
    }  
    printf ("\n");
  }
}

